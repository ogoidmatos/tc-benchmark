#include <hip/hip_runtime.h>

#include <driver_functions.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>

#define M 16
#define N 8
#define K 256

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 32768L / 4
#define A_SIZE M *K *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define B_SIZE K *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define C_SIZE M *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define ITERATIONS 32768

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

// Kernel function
__global__ void benchmark_alt(int *d_A, int *d_B, int *d_C,
                              uint64_t *d_startClk, uint64_t *d_stopClk,
                              uint64_t *d_timeStart, uint64_t *d_timeStop) {
  // Code to be executed on the GPU
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint64_t time_start = 0;
  uint64_t time_stop = 0;

  // create registers for threads
  int fragsA[4];
  int fragsB[2];
  int fragsC[4];

  for (int i = 0; i < 2; i++) {
    fragsB[i] = d_B[i + id * 2];
  }
  for (int i = 0; i < 4; i++) {
    fragsA[i] = d_B[i + id * 4];
    fragsC[i] = d_C[i + id * 4];
  }

  // uint32_t const *A = reinterpret_cast<uint32_t const *>(
  //     &fragsA[0]);  // change from half to bit 32 which is what the mma takes
  // uint32_t const *B = reinterpret_cast<uint32_t const *>(&fragsB[0]);
  // float *C = reinterpret_cast<float *>(&fragsC[0]);

  // synchronize threads
  asm volatile("bar.sync 0;");

  // start timing
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_start)::"memory");
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  for (int i = 0; i < ITERATIONS; i++) {
    // assembly mma
    asm volatile(
        "mma.sync.aligned.m16n8k256.row.col.s32.b1.b1.s32.xor.popc "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
        : "+r"(fragsC[0]), "+r"(fragsC[1]), "+r"(fragsC[2]), "+r"(fragsC[3])
        : "r"(fragsA[0]), "r"(fragsA[1]), "r"(fragsA[2]), "r"(fragsA[3]),
          "r"(fragsB[0]), "r"(fragsB[1]));
    //__syncwarp();
  }
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_stop)::"memory");

  for (int i = 0; i < 4; i++) {
    d_C[i + id * 4] = fragsC[i];
  }

  d_startClk[id] = start;
  d_stopClk[id] = stop;
  d_timeStart[id] = time_start;
  d_timeStop[id] = time_stop;
}

// D = A*B + D
int main() {
  // Code to be executed on the CPU

  // Print CUDA info
  printCudaInfo();

  // Calculate matrix dimensions
  long dimA = A_SIZE;
  int dimB = B_SIZE;
  int dimC = C_SIZE;  // dimC is the same as dimD

  // Allocate host memory
  int *h_A = (int *)malloc(dimA * sizeof(int));
  int *h_B = (int *)malloc(dimB * sizeof(int));
  int *h_C = (int *)malloc(dimC * sizeof(int));

  // Initialize host memory
  for (int i = 0; i < dimA; i++) {
    h_A[i] = 0.0f;
  }
  for (int i = 0; i < dimB; i++) {
    h_B[i] = 0.0f;
  }
  for (int i = 0; i < dimC; i++) {
    h_C[i] = 0.0f;
  }

  // Allocate device memory
  int *d_A, *d_B;
  int *d_C;
  cudaCheckError(hipMalloc((void **)&d_A, dimA * sizeof(int)));
  cudaCheckError(hipMalloc((void **)&d_B, dimB * sizeof(int)));
  cudaCheckError(hipMalloc((void **)&d_C, dimC * sizeof(int)));

  // Copy host memory to device
  cudaCheckError(
      hipMemcpy(d_A, h_A, dimA * sizeof(int), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_B, h_B, dimB * sizeof(int), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_C, h_C, dimC * sizeof(int), hipMemcpyHostToDevice));

  // handle clock
  uint64_t *startClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_startClk, *d_stopClk;
  cudaCheckError(hipMalloc((void **)&d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // handle timings
  uint64_t *timeStart =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopStop =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_timeStart, *d_timeStop;
  cudaCheckError(hipMalloc((void **)&d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // Launch kernel on the GPU
  benchmark_alt<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
      d_A, d_B, d_C, d_startClk, d_stopClk, d_timeStart, d_timeStop);

  // Wait for GPU to finish
  cudaCheckError(hipDeviceSynchronize());

  // Copy device memory to host
  cudaCheckError(hipMemcpy(startClk, d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopClk, d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(timeStart, d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopStop, d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));

  cudaCheckError(hipDeviceSynchronize());

  uint64_t total_clk =
      *std::max_element(stopClk, stopClk + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(startClk, startClk + NUM_BLOCKS * THREADS_PER_BLOCK);
  double total_time =
      *std::max_element(stopStop, stopStop + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(timeStart, timeStart + NUM_BLOCKS * THREADS_PER_BLOCK);

  total_time = total_time / 1e9;

  uint64_t fma =
      (uint64_t)M * N * K * ITERATIONS * (THREADS_PER_BLOCK / 32) * NUM_BLOCKS;
  float bw = (float)fma / (float)total_clk;

  double FLOPS = fma * 2 / total_time / 1e12;

  std::cout
      << "mma.sync.aligned.m16n8k256.row.col.s32.b1.b1.s32.xor.popc  latency "
      << (float)total_clk / (float)ITERATIONS << " cycles\n";
  std::cout
      << "mma.sync.aligned.m16n8k256.row.col.s32.b1.b1.s32.xor.popc  FMA Count "
      << fma << "\n";
  std::cout << "FMA tensor bandwidth = " << bw << " (FMA/clk/SM)\n";

  std::cout << "Total Clk number = " << total_clk << "\n";

  std::cout << "Total Time number = " << total_time << " (sec)\n";
  std::cout << "FLOPS = " << FLOPS << "(TFLOPs) \n";

  // Free device memory
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));
  cudaCheckError(hipFree(d_startClk));
  cudaCheckError(hipFree(d_stopClk));
  cudaCheckError(hipFree(d_timeStart));
  cudaCheckError(hipFree(d_timeStop));

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(startClk);
  free(stopClk);
  free(timeStart);
  free(stopStop);

  return 0;
}
