#include <hip/hip_runtime.h>

#include <driver_functions.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>

#define M 8
#define N 8
#define K 4

#define THREADS_PER_BLOCK 1024
#define A_SIZE M *K *(THREADS_PER_BLOCK / 32)
#define B_SIZE K *N *(THREADS_PER_BLOCK / 32)
#define C_SIZE M *N *(THREADS_PER_BLOCK / 32)
#define ITERATIONS 32768

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

// Kernel function
__global__ void benchmark_alt(half *d_A, half *d_B, float *d_C,
                              uint64_t *d_startClk, uint64_t *d_stopClk) {
  // Code to be executed on the GPU
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t start = 0;
  uint64_t stop = 0;

  // create registers for threads
  half fragsA[4];
  half fragsB[4];
  float fragsC[8];

  for (int i = 0; i < 8; i++) {
    fragsC[i] = d_C[i + id * 8];
  }
  for (int i = 0; i < 4; i++) {
    fragsB[i] = d_B[i + id * 4];
    fragsA[i] = d_A[i + id * 4];
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(
      &fragsA[0]);  // change from half to bit 32 which is what the mma takes
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&fragsB[0]);
  float *C = reinterpret_cast<float *>(&fragsC[0]);

  // synchronize threads
  asm volatile("bar.sync 0;");

  // start timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  for (int i = 0; i < ITERATIONS; i++) {
    // assembly mma
    asm volatile(
        "mma.sync.aligned.m8n8k4.row.col.f32.f16.f16.f32 "
        "{%0,%1,%2,%3,%4,%5,%6,%7}, {%8,%9}, {%10,%11}, "
        "{%0,%1,%2,%3,%4,%5,%6,%7};\n"
        : "+f"(C[0]), "+f"(C[1]), "+f"(C[2]), "+f"(C[3]), "+f"(C[4]),
          "+f"(C[5]), "+f"(C[6]), "+f"(C[7])
        : "r"(A[0]), "r"(A[1]), "r"(B[0]), "r"(B[1]));
    //__syncwarp();
  }
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");

  for (int i = 0; i < 8; i++) {
    d_C[i + id * 8] = fragsC[i];
  }

  d_startClk[id] = start;
  d_stopClk[id] = stop;
}

// D = A*B + D
int main() {
  // Code to be executed on the CPU

  // Print CUDA info
  printCudaInfo();

  // Calculate matrix dimensions
  int BLOCKS = 1;
  int dimA = A_SIZE;
  int dimB = B_SIZE;
  int dimC = C_SIZE;  // dimC is the same as dimD

  // Allocate host memory
  half *h_A = (half *)malloc(dimA * sizeof(half));
  half *h_B = (half *)malloc(dimB * sizeof(half));
  float *h_C = (float *)malloc(dimC * sizeof(float));

  // Initialize host memory
  for (int i = 0; i < dimA; i++) {
    h_A[i] = 0.0f;
  }
  for (int i = 0; i < dimB; i++) {
    h_B[i] = 0.0f;
  }
  for (int i = 0; i < dimC; i++) {
    h_C[i] = 0.0f;
  }

  // Allocate device memory
  half *d_A, *d_B;
  float *d_C;
  cudaCheckError(hipMalloc((void **)&d_A, dimA * sizeof(half)));
  cudaCheckError(hipMalloc((void **)&d_B, dimB * sizeof(half)));
  cudaCheckError(hipMalloc((void **)&d_C, dimC * sizeof(float)));

  // Copy host memory to device
  cudaCheckError(
      hipMemcpy(d_A, h_A, dimA * sizeof(half), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_B, h_B, dimB * sizeof(half), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_C, h_C, dimC * sizeof(float), hipMemcpyHostToDevice));

  // handle clock
  uint64_t *startClk = (uint64_t *)malloc(THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopClk = (uint64_t *)malloc(THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_startClk, *d_stopClk;
  cudaCheckError(
      hipMalloc((void **)&d_startClk, THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(
      hipMalloc((void **)&d_stopClk, THREADS_PER_BLOCK * sizeof(uint64_t)));

  // Launch kernel on the GPU
  benchmark_alt<<<BLOCKS, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, d_startClk,
                                               d_stopClk);

  // Wait for GPU to finish
  cudaCheckError(hipDeviceSynchronize());

  // Copy device memory to host
  cudaCheckError(hipMemcpy(startClk, d_startClk,
                            THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopClk, d_stopClk,
                            THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));

  cudaCheckError(hipDeviceSynchronize());

  uint64_t total_time =
      *std::max_element(stopClk, stopClk + THREADS_PER_BLOCK) -
      *std::min_element(startClk, startClk + THREADS_PER_BLOCK);

  uint64_t fma = (uint64_t)M * N * K * ITERATIONS * (THREADS_PER_BLOCK / 32);
  float bw = (float)fma / (float)total_time;

  std::cout << "mma.sync.aligned.m8n8k4.row.col.f32.f16.f16.f32  latency "
            << (float)total_time / (float)ITERATIONS << " cycles\n";
  std::cout << "mma.sync.aligned.m8n8k4.row.col.f32.f16.f16.f32  FMA Count "
            << fma << "\n";
  std::cout << "FMA tensor bandwidth = " << bw << " (FMA/clk/SM)\n";

  std::cout << "Total Clk number = " << total_time << "\n";

  // Free device memory
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));
  cudaCheckError(hipFree(d_startClk));
  cudaCheckError(hipFree(d_stopClk));

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(startClk);
  free(stopClk);

  return 0;
}
