#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include "../../nvml_tools.cu"

#define M 16
#define N 8
#define K 16

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 32768
#define A_SIZE M *K *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define B_SIZE K *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define C_SIZE M *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define ITERATIONS 32768
// stride required to force all the data to come from DRAM
#define STRIDE 32768 * 4L

#define FLOP 2

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

// Kernel function
__global__ void benchmark_alt(float *d_A, float *d_B, float *d_C, float *d_X,
                              float *d_Y, uint64_t *d_startClk,
                              uint64_t *d_stopClk, uint64_t *d_timeStart,
                              uint64_t *d_timeStop) {
  // Code to be executed on the GPU
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint64_t time_start = 0;
  uint64_t time_stop = 0;

  // create registers for threads
  float fragsA[4];
  float fragsB[2];
  float fragsC[2];

  for (int i = 0; i < 4; i++) {
    fragsA[i] = d_A[i + id * 4];
  }
  for (int i = 0; i < 2; i++) {
    fragsB[i] = d_B[i + id * 2];
    fragsC[i] = d_C[i + id * 2];
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(
      &fragsA[0]);  // change from half to bit 32 which is what the mma takes
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&fragsB[0]);
  uint32_t *C = reinterpret_cast<uint32_t *>(&fragsC[0]);

  // start timing
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_start)::"memory");
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  for (int i = 0; i < ITERATIONS; i += 32) {
#pragma unroll
    for (int j = 0; j < 32; j++) {
      // SWITCH ORDER HERE TO MAKE SURE THAT C0 IS WRITTEN TO MEMORY BEFORE
      // BEING OVERWRITTEN; NECESSARY FOR COMPILER TO NOT OPTIMIZE AWAY THE
      // COMPUTATION
      d_Y[id + j * STRIDE] = fragsC[0];
      fragsC[0] = d_X[id + j * STRIDE];

#pragma unroll
      for (int j = 0; j < FLOP; j++) {
        // assembly mma
        asm volatile(
            "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 "
            "{%0,%1}, {%2,%3,%4,%5}, {%6,%7}, {%0,%1};\n"
            : "+r"(C[0]), "+r"(C[1])
            : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1]));
      }
    }
  }
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_stop)::"memory");

  for (int i = 0; i < 2; i++) {
    d_C[i + id * 2] = fragsC[i];
  }

  d_startClk[id] = start;
  d_stopClk[id] = stop;
  d_timeStart[id] = time_start;
  d_timeStop[id] = time_stop;
}

// D = A*B + D
int main() {
  // Code to be executed on the CPU
  // start nvml
  // thread to measure power configuration
  std::thread measuring_thread;
  monitor_args thread_args;
  thread_args.powerArray = std::vector<int>();
  thread_args.clockArray = std::vector<int>();
  thread_args.flag = 0;

  init_nvml(&thread_args, &measuring_thread);
  cudaCheckError(hipDeviceSynchronize());

  // Print CUDA info
  printCudaInfo();

  // Calculate matrix dimensions
  int dimA = A_SIZE;
  int dimB = B_SIZE;
  int dimC = C_SIZE;  // dimC is the same as dimD

  // Allocate host memory
  float *h_A = (float *)malloc(dimA * sizeof(float));
  float *h_B = (float *)malloc(dimB * sizeof(float));
  float *h_C = (float *)malloc(dimC * sizeof(float));

  // Initialize host memory
  for (int i = 0; i < dimA; i++) {
    h_A[i] = 0.0f;
  }
  for (int i = 0; i < dimB; i++) {
    h_B[i] = 0.0f;
  }
  for (int i = 0; i < dimC; i++) {
    h_C[i] = 0.0f;
  }

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  cudaCheckError(hipMalloc((void **)&d_A, dimA * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_B, dimB * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_C, dimC * sizeof(float)));

  // Copy host memory to device
  cudaCheckError(
      hipMemcpy(d_A, h_A, dimA * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_B, h_B, dimB * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_C, h_C, dimC * sizeof(float), hipMemcpyHostToDevice));

  // handle l2 mem handles
  float *h_X = (float *)malloc((NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) *
                               sizeof(float));
  float *h_Y = (float *)malloc((NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) *
                               sizeof(float));
  float *d_X, *d_Y;
  cudaCheckError(hipMalloc(
      (void **)&d_X,
      (NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) * sizeof(float)));
  cudaCheckError(hipMalloc(
      (void **)&d_Y,
      (NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) * sizeof(float)));
  cudaCheckError(hipMemcpy(
      d_X, h_X, (NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) * sizeof(float),
      hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(
      d_Y, h_Y, (NUM_BLOCKS * THREADS_PER_BLOCK + 32 * STRIDE) * sizeof(float),
      hipMemcpyHostToDevice));

  // handle clock
  uint64_t *startClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_startClk, *d_stopClk;
  cudaCheckError(hipMalloc((void **)&d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // handle timings
  uint64_t *timeStart =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopStop =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_timeStart, *d_timeStop;
  cudaCheckError(hipMalloc((void **)&d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  thread_args.flag = 1;
  // Launch kernel on the GPU
  benchmark_alt<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
      d_A, d_B, d_C, d_X, d_Y, d_startClk, d_stopClk, d_timeStart, d_timeStop);

  // Wait for GPU to finish
  cudaCheckError(hipDeviceSynchronize());
  thread_args.flag = 0;
  stop_nvml(&measuring_thread, thread_args.powerArray, thread_args.clockArray);

  // Copy device memory to host
  cudaCheckError(hipMemcpy(startClk, d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopClk, d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(timeStart, d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopStop, d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));

  cudaCheckError(hipDeviceSynchronize());

  uint64_t total_clk =
      *std::max_element(stopClk, stopClk + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(startClk, startClk + NUM_BLOCKS * THREADS_PER_BLOCK);
  double total_time =
      *std::max_element(stopStop, stopStop + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(timeStart, timeStart + NUM_BLOCKS * THREADS_PER_BLOCK);

  total_time = total_time / 1e9;

  uint64_t fma = (uint64_t)M * N * K * ITERATIONS * (THREADS_PER_BLOCK / 32) *
                 NUM_BLOCKS * FLOP;

  long bytes = sizeof(float) * 2 * ITERATIONS * THREADS_PER_BLOCK *
               NUM_BLOCKS;  // 2 for read and write

  double bw = (float)bytes / (float)total_time / 1e9;

  double FLOPS = fma * 2 / total_time / 1e12;

  std::cout << "Bandwidth = " << bw << " (bytes/sec)\n";

  std::cout << "FLOPS = " << FLOPS << "(TFLOPs) \n";
  std::cout << "AI = " << fma * 2. / bytes << " (FLOP/byte)\n";

  std::cout << "Total Clk number = " << total_clk << "\n";

  std::cout << "Total Time number = " << total_time << " (sec)\n";
  std::cout << "Average Clock Frequency = " << total_clk / total_time / 1e6
            << " (MHz)\n";

  std::cout << "---------------------------------------------------------\n";

  // Free device memory
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));
  cudaCheckError(hipFree(d_X));
  cudaCheckError(hipFree(d_Y));
  cudaCheckError(hipFree(d_startClk));
  cudaCheckError(hipFree(d_stopClk));
  cudaCheckError(hipFree(d_timeStart));
  cudaCheckError(hipFree(d_timeStop));

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_X);
  free(h_Y);
  free(startClk);
  free(stopClk);
  free(timeStart);
  free(stopStop);

  return 0;
}
