#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cstdlib>
#include <iostream>

#include "../../nvml_tools.cu"

#define M 16
#define N 8
#define K 8

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 32768
#define A_SIZE M *K *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define B_SIZE K *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define C_SIZE M *N *(THREADS_PER_BLOCK / 32) * NUM_BLOCKS
#define ITERATIONS 32768 / 2

#define MEM 1
#define FLOP 16

#if MEM > FLOP
#define REMAINDER MEM - FLOP
#elif FLOP > MEM
#define REMAINDER FLOP - MEM
#endif

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

// Kernel function
__global__ void benchmark_alt(float *d_A, float *d_B, float *d_C, float *d_X,
                              float *d_Y, uint64_t *d_startClk,
                              uint64_t *d_stopClk, uint64_t *d_timeStart,
                              uint64_t *d_timeStop) {
  // Code to be executed on the GPU
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint64_t time_start = 0;
  uint64_t time_stop = 0;

  // create registers for threads
  float fragsA[4];
  float fragsB[2];
  float fragsC[4];

  for (int i = 0; i < 4; i++) {
    fragsA[i] = d_A[i + id * 4];
    fragsC[i] = d_C[i + id * 4];
  }
  for (int i = 0; i < 2; i++) {
    fragsB[i] = d_B[i + id * 2];
  }

  uint32_t const *A = reinterpret_cast<uint32_t const *>(
      &fragsA[0]);  // change from half to bit 32 which is what the mma takes
  uint32_t const *B = reinterpret_cast<uint32_t const *>(&fragsB[0]);
  float *C = reinterpret_cast<float *>(&fragsC[0]);

  // start timing
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_start)::"memory");
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  for (int i = 0; i < ITERATIONS; i++) {
#if MEM == 1 || FLOP == 1
#pragma unroll
    for (int j = 0; j < MEM; j++) {
      // SWITCH ORDER HERE TO MAKE SURE THAT C0 IS WRITTEN TO MEMORY BEFORE
      // BEING OVERWRITTEN; NECESSARY FOR COMPILER TO NOT OPTIMIZE AWAY THE
      // COMPUTATION
      d_Y[id] = fragsC[0];
      fragsC[0] = d_X[id];
    }
#pragma unroll
    for (int j = 0; j < FLOP; j++) {
      // assembly mma
      asm volatile(
          "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 "
          "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
          : "+f"(C[0]), "+f"(C[1]), "+f"(C[2]), "+f"(C[3])
          : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1]));
    }
#elif MEM > FLOP
#pragma unroll
    for (int j = 0; j < FLOP; j++) {
      d_Y[id] = fragsC[0];
      fragsC[0] = d_X[id];
      // assembly mma
      asm volatile(
          "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 "
          "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
          : "+f"(C[0]), "+f"(C[1]), "+f"(C[2]), "+f"(C[3])
          : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1]));
    }
#pragma unroll
    for (int j = 0; j < REMAINDER; j++) {
      d_Y[id] = fragsC[0];
      fragsC[0] = d_X[id];
    }
#else
#pragma unroll
    for (int j = 0; j < MEM; j++) {
      d_Y[id] = fragsC[0];
      fragsC[0] = d_X[id];
      // assembly mma
      asm volatile(
          "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 "
          "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
          : "+f"(C[0]), "+f"(C[1]), "+f"(C[2]), "+f"(C[3])
          : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1]));
    }
#pragma unroll
    for (int j = 0; j < REMAINDER; j++) {
      asm volatile(
          "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 "
          "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
          : "+f"(C[0]), "+f"(C[1]), "+f"(C[2]), "+f"(C[3])
          : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1]));
    }
#endif
  }
  // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_stop)::"memory");

  for (int i = 0; i < 4; i++) {
    d_C[i + id * 4] = fragsC[i];
  }

  d_startClk[id] = start;
  d_stopClk[id] = stop;
  d_timeStart[id] = time_start;
  d_timeStop[id] = time_stop;
  // d_X[id] = fragsC[0];
}

// D = A*B + D
int main() {
  // Code to be executed on the CPU
  // start nvml
  // thread to measure power configuration
  std::thread measuring_thread;
  monitor_args thread_args;
  thread_args.powerArray = std::vector<int>();
  thread_args.clockArray = std::vector<int>();
  thread_args.flag = 0;

  init_nvml(&thread_args, &measuring_thread, false);
  cudaCheckError(hipDeviceSynchronize());

  // Print CUDA info
  printCudaInfo();

  // Calculate matrix dimensions
  int dimA = A_SIZE;
  int dimB = B_SIZE;
  int dimC = C_SIZE;  // dimC is the same as dimD

  // Allocate host memory
  float *h_A = (float *)malloc(dimA * sizeof(float));
  float *h_B = (float *)malloc(dimB * sizeof(float));
  float *h_C = (float *)malloc(dimC * sizeof(float));

  // Initialize host memory
  for (int i = 0; i < dimA; i++) {
    h_A[i] = 0.0f;
  }
  for (int i = 0; i < dimB; i++) {
    h_B[i] = 0.0f;
  }
  for (int i = 0; i < dimC; i++) {
    h_C[i] = 0.0f;
  }

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  cudaCheckError(hipMalloc((void **)&d_A, dimA * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_B, dimB * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_C, dimC * sizeof(float)));

  // Copy host memory to device
  cudaCheckError(
      hipMemcpy(d_A, h_A, dimA * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_B, h_B, dimB * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(
      hipMemcpy(d_C, h_C, dimC * sizeof(float), hipMemcpyHostToDevice));

  // handle l2 mem handles
  float *h_X = (float *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float));
  float *h_Y = (float *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float));
  float *d_X, *d_Y;
  cudaCheckError(hipMalloc((void **)&d_X,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_Y,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float)));
  cudaCheckError(hipMemcpy(d_X, h_X,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float),
                            hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_Y, h_Y,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float),
                            hipMemcpyHostToDevice));

  // handle clock
  uint64_t *startClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_startClk, *d_stopClk;
  cudaCheckError(hipMalloc((void **)&d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // handle timings
  uint64_t *timeStart =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopStop =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_timeStart, *d_timeStop;
  cudaCheckError(hipMalloc((void **)&d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // Prepare
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Start record
  hipEventRecord(start, 0);

  thread_args.flag = 1;
  // Launch kernel on the GPU
  benchmark_alt<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
      d_A, d_B, d_C, d_X, d_Y, d_startClk, d_stopClk, d_timeStart, d_timeStop);

  // Stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);  // that's our time!
  printf("Elapsed time: %f ms\n", elapsedTime);
  // Clean up:
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Wait for GPU to finish
  cudaCheckError(hipDeviceSynchronize());
  thread_args.flag = 0;
  stop_nvml(&measuring_thread, thread_args.powerArray, thread_args.clockArray);

  // Copy device memory to host
  cudaCheckError(hipMemcpy(startClk, d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopClk, d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(timeStart, d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopStop, d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));

  cudaCheckError(hipDeviceSynchronize());

  uint64_t total_clk =
      *std::max_element(stopClk, stopClk + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(startClk, startClk + NUM_BLOCKS * THREADS_PER_BLOCK);
  double total_time =
      *std::max_element(stopStop, stopStop + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(timeStart, timeStart + NUM_BLOCKS * THREADS_PER_BLOCK);

  total_time = total_time / 1e9;

  uint64_t fma = (uint64_t)M * N * K * ITERATIONS * (THREADS_PER_BLOCK / 32) *
                 NUM_BLOCKS * FLOP;

  long bytes = sizeof(float) * 2 * ITERATIONS * THREADS_PER_BLOCK * NUM_BLOCKS *
               MEM;  // 2 for read and write

  double bw = (float)bytes / (float)total_time / 1e9;

  double FLOPS = fma * 2 / total_time / 1e12;

  std::cout << "Bandwidth = " << bw << " (bytes/sec)\n";

  std::cout << "FLOPS = " << FLOPS << "(TFLOPs) \n";
  std::cout << "AI = " << fma * 2. / bytes << " (FLOP/byte)\n";

  std::cout << "Total Clk number = " << total_clk << "\n";

  std::cout << "Total Time number = " << total_time << " (sec)\n";
  std::cout << "Average Clock Frequency = " << total_clk / total_time / 1e6
            << " (MHz)\n";
  std::cout << bytes << " bytes\n";

  std::cout << "---------------------------------------------------------\n";

  // Free device memory
  cudaCheckError(hipFree(d_A));
  cudaCheckError(hipFree(d_B));
  cudaCheckError(hipFree(d_C));
  cudaCheckError(hipFree(d_X));
  cudaCheckError(hipFree(d_Y));
  cudaCheckError(hipFree(d_startClk));
  cudaCheckError(hipFree(d_stopClk));
  cudaCheckError(hipFree(d_timeStart));
  cudaCheckError(hipFree(d_timeStop));

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_X);
  free(h_Y);
  free(startClk);
  free(stopClk);
  free(timeStart);
  free(stopStop);

  return 0;
}
