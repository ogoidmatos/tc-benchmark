#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <cstdlib>
#include <iostream>
#include <vector>

#include "../../nvml_tools.cu"

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 32768L
#define ITERATIONS 32768L
#define MEM 2
#define FLOP 256
#define AI ((float)FLOP / MEM)

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

void printCudaInfo() {
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    printf("   Clock:      %.2f MHz\n", (deviceProps.clockRate / 1000.0f));
  }
  printf("---------------------------------------------------------\n");
}

// Kernel function
template <class T>
__global__ void benchmark_alt(T *d_X, T *d_Y, uint64_t *d_startClk,
                              uint64_t *d_stopClk, uint64_t *d_timeStart,
                              uint64_t *d_timeStop) {
  // Code to be executed on the GPU
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t start = 0;
  uint64_t stop = 0;
  uint64_t time_start = 0;
  uint64_t time_stop = 0;

  T a = (T)id;
  T b = a + 1;
  T c = b + 1;
  T d = c + 1;
  // synchronize threads
  // asm volatile("bar.sync 0;");

  // start timing
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_start)::"memory");
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");

  // #pragma unroll
  for (int i = 0; i < ITERATIONS; i++) {
//  assembly mma
#pragma unroll
    for (int j = 0; j < MEM; j++) {
      a = d_X[id];
      d_Y[id] = a;
    }
#pragma unroll
    for (int j = 0; j < FLOP; j++) {
      a = a * a + b;
      b = b * b + c;
      c = c * c + d;
      d = d * d + a;
    }
  }

  // // stop timing
  asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
  asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(time_stop)::"memory");

  d_startClk[id] = start;
  d_stopClk[id] = stop;
  d_timeStart[id] = time_start;
  d_timeStop[id] = time_stop;
  d_X[id] = d;
}

// D = A*B + D
int main() {
  // Code to be executed on the CPU

  // start nvml
  // thread to measure power configuration
  std::thread measuring_thread;
  monitor_args thread_args;
  thread_args.powerArray = std::vector<int>();
  thread_args.clockArray = std::vector<int>();
  thread_args.flag = 0;

  init_nvml(&thread_args, &measuring_thread);
  cudaCheckError(hipDeviceSynchronize());

  // Print CUDA info
  printCudaInfo();

  float *h_X = (float *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float));
  float *h_Y = (float *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float));
  float *d_X, *d_Y;
  cudaCheckError(hipMalloc((void **)&d_X,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float)));
  cudaCheckError(hipMemcpy(d_X, h_X,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float),
                            hipMemcpyHostToDevice));
  cudaCheckError(hipMalloc((void **)&d_Y,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float)));
  cudaCheckError(hipMemcpy(d_Y, h_Y,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(float),
                            hipMemcpyHostToDevice));

  // handle clock
  uint64_t *startClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopClk =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_startClk, *d_stopClk;
  cudaCheckError(hipMalloc((void **)&d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  // handle timings
  uint64_t *timeStart =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));
  uint64_t *stopStop =
      (uint64_t *)malloc(NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t));

  uint64_t *d_timeStart, *d_timeStop;
  cudaCheckError(hipMalloc((void **)&d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));
  cudaCheckError(hipMalloc((void **)&d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t)));

  thread_args.flag = 1;
  // Launch kernel on the GPU
  benchmark_alt<float><<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
      d_X, d_Y, d_startClk, d_stopClk, d_timeStart, d_timeStop);

  // Wait for GPU to finish
  cudaCheckError(hipDeviceSynchronize());
  thread_args.flag = 0;
  stop_nvml(&measuring_thread, thread_args.powerArray, thread_args.clockArray);

  // Copy device memory to host
  cudaCheckError(hipMemcpy(startClk, d_startClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopClk, d_stopClk,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(timeStart, d_timeStart,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(stopStop, d_timeStop,
                            NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(uint64_t),
                            hipMemcpyDeviceToHost));

  cudaCheckError(hipDeviceSynchronize());

  uint64_t total_clk =
      *std::max_element(stopClk, stopClk + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(startClk, startClk + NUM_BLOCKS * THREADS_PER_BLOCK);
  double total_time =
      *std::max_element(stopStop, stopStop + NUM_BLOCKS * THREADS_PER_BLOCK) -
      *std::min_element(timeStart, timeStart + NUM_BLOCKS * THREADS_PER_BLOCK);

  total_time = total_time / 1e9;

  long fma = 4 * ITERATIONS * THREADS_PER_BLOCK * NUM_BLOCKS *
             FLOP;  // 4 fma instructions, 4*2 flops

  long bytes = sizeof(float) * 2 * ITERATIONS * THREADS_PER_BLOCK * NUM_BLOCKS *
               MEM;  // 2 for read and write

  // float fma_bw = (float)fma / (float)total_clk;

  double FLOPS = fma * 2 / total_time / 1e12;

  double bw = (float)bytes / (float)total_time / 1e9;

  // std::cout << "FMA tensor bandwidth = " << bw << " (FMA/clk/SM)\n";
  std::cout << "Bandwidth = " << bw << " (GB/s)\n";
  std::cout << "FLOPS = " << FLOPS << "(TFLOPs) \n";
  std::cout << "AI = " << AI << " (FLOP/byte)\n";

  std::cout << "Total Clk number = " << total_clk << "\n";

  std::cout << "Total Time number = " << total_time << " (sec)\n";
  std::cout << "Average Clock Frequency = " << total_clk / total_time / 1e6
            << " (MHz)\n";

  // std::cout << "---------------------------------------------------------\n";

  // Free device memory

  cudaCheckError(hipFree(d_startClk));
  cudaCheckError(hipFree(d_stopClk));
  cudaCheckError(hipFree(d_timeStart));
  cudaCheckError(hipFree(d_timeStop));
  cudaCheckError(hipFree(d_X));
  cudaCheckError(hipFree(d_Y));

  // Free host memory

  free(startClk);
  free(stopClk);
  free(timeStart);
  free(stopStop);
  free(h_X);
  free(h_Y);

  return 0;
}
